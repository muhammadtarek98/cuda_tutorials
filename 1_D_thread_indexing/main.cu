#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

__global__ void thread_1D_indexing(int *a)
{
    int thread_id=threadIdx.x+blockIdx.x*blockDim.x;

    printf("thread index: %d value: %d\n",thread_id,a[thread_id]);
}

int main()
{
    int n=8,nbytes=sizeof(int)*n;
    int h_arr[]{1,2,3,4,5,6,7,8};
    int *d_arr;
    hipMalloc((void**)&d_arr,nbytes);
    hipMemcpy(d_arr,h_arr,nbytes,hipMemcpyHostToDevice);
    dim3 block(n/2);
    dim3 grid(2);
    thread_1D_indexing<<<grid,block>>>(d_arr);
    hipDeviceSynchronize();
    hipFree(d_arr);
    hipDeviceReset();

    return 0;
}