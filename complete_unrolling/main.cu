#include<bits/stdc++.h>

#include <hip/hip_runtime.h>
void seq_array_accum(int &a, const std::vector<int> &arr)
{
    for (const auto &i : arr)
    {
        a+=i;
    }
}
void init(std::vector<int>&arr)
{
    for (int i=0;i<static_cast<int>(arr.size());++i)
    {
        arr[i]=i%10;
    }
}
__global__ void kernel(int *input,int *part,int size){
    auto tid=threadIdx.x;
    auto index=tid+blockIdx.x*blockDim.x;
    int *window=input+blockDim.x*blockIdx.x;
    if(blockDim.x>=1024&&tid<512){
    window[tid]+=window[tid+512];
    }
    __syncthreads();
    if(blockDim.x>=512&&tid<256){
    window[tid]+=window[tid+256];
    }
    __syncthreads();
    if(blockDim.x>=256&&tid<128){
        window[tid]+=window[tid+128];
    }
    __syncthreads();
    if(blockDim.x>=128&&tid<64){
        window[tid]+=window[tid+64];
    }
    __syncthreads();
    if(tid<32){
        volatile int *vsmem=window;
        vsmem[tid]+=vsmem[tid+32];
        vsmem[tid]+=vsmem[tid+16];
        vsmem[tid]+=vsmem[tid+8];
        vsmem[tid]+=vsmem[tid+4];
        vsmem[tid]+=vsmem[tid+2];
        vsmem[tid]+=vsmem[tid+1];
    }
    __syncthreads();

    if(tid==0){
    part[blockIdx.x]=window[0];
    }
    __syncthreads();

}
int main(int argc,const char *argv [])
{
    int size=1<<27;
    int block_size=1024;
    int cpu_res=0,gpu_res=0;
    dim3 blocks(block_size);
    dim3 grid(size/block_size);
    const auto input_byte_size=size*sizeof(int);
    const auto part_byte_size=grid.x*sizeof(int);
    std::vector<int> h_input(size);
    std::vector<int>h_part(grid.x,0);
    int *d_input=nullptr;
    int *d_part=nullptr;
    init(h_input);
    seq_array_accum(cpu_res,h_input);
    hipMalloc(reinterpret_cast<void**>(&d_input),input_byte_size);
    hipMalloc(reinterpret_cast<void**>(&d_part),part_byte_size);
    hipMemset(d_part,0,part_byte_size);
    hipMemcpy(d_input,h_input.data(),input_byte_size,hipMemcpyHostToDevice);
    kernel<<<grid,blocks>>>(d_input,d_part,size);
    hipDeviceSynchronize();
    hipMemcpy(h_part.data(),d_part,part_byte_size,hipMemcpyDeviceToHost);
    seq_array_accum(gpu_res,h_part);
    std::cout<<gpu_res<<" "<<cpu_res<<"\n";
    std::cout<<(gpu_res==cpu_res)<<"\n";
    h_input.clear(),h_part.clear(),hipFree(d_part),hipFree(d_input);
    hipDeviceReset();


    return 0;
}