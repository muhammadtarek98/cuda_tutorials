#include <bits/stdc++.h>

#include <hip/hip_runtime.h>

int main(int argc,const char *argv [])
{
    int isize=1<<25;
    int nbyes=isize* sizeof(float );
    float *h_a,*d_a;
    hipHostMalloc(reinterpret_cast<float **>(&h_a),nbyes, hipHostMallocDefault);
    hipMalloc(reinterpret_cast<float **>(&d_a),nbyes);
    for(int i=0;i<isize;++i){
        h_a[i]=5;
    }
    hipMemcpy(d_a,h_a,nbyes,hipMemcpyHostToDevice);
    hipMemcpy(h_a,d_a,nbyes,hipMemcpyDeviceToHost);
    hipFree(d_a), hipHostFree(h_a);

    return 0;
}