#include <bits/stdc++.h>

#include <hip/hip_runtime.h>

int main(int argc,const char *argv [])
{
    int isize=1<<25;
    auto nbyes=isize* sizeof(float);
    std::vector<float> h_a(isize);std::vector<float>d_a(isize);
    hipHostMalloc(reinterpret_cast<float **>(h_a.data()),nbyes, hipHostMallocDefault);
    hipMalloc(reinterpret_cast<float **>(d_a.data()),nbyes);
    for(int i=0;i<isize;++i){
        h_a[i]=5;
    }
    hipMemcpy(d_a.data(),h_a.data(),nbyes,hipMemcpyHostToDevice);
    hipMemcpy(h_a.data(),d_a.data(),nbyes,hipMemcpyDeviceToHost);
    hipFree(d_a.data()), hipHostFree(h_a.data());

    return 0;
}