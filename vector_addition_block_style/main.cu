
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#define  H2D cudaMemcpyHosttoDevice
#define  D2H cudaMemcpyDevicetoHost

__global__ void array_addition(int *a, int *b, int *c)
{
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}
void random_ints(int* arr, int size) {
    // Seed the random number generator
    srand(time(NULL));

    // Fill the array with random integers
    for(int i = 0; i < size; i++) {
        arr[i] = rand() % 100;  // Generate random numbers between 0 and 99
    }
}
int main()
{

    int *a,*b,*c;
    int *d_a,*d_b,*d_c;
    int N = 512;
    int size=N*sizeof(int);
    a=(int*)malloc(size);
    b=(int*)malloc(size);
    c=(int*)malloc(size);
    random_ints(a,N);
    random_ints(b,N);
    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&d_b,size);
    hipMalloc((void**)&d_c,size);
    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
    array_addition<<<N,1>>>(d_a,d_b,d_c);
    hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
    for(int i=0;i<N;i++)
    {
        std::cout<<c[i]<<" ";
    }
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}