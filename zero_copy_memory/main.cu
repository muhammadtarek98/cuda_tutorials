#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
void init(std::vector<int>&arr)
{
    for (int i=0;i<(int)arr.size();++i)
    {
        arr[i]=i%10;
    }
}
void init(int *arr,int sz)
{
    for (int i=0;i<sz;++i)
    {
        arr[i]=i%10;
    }
}
__global__ void kernel(int *a,int *b,int *c,const int n)
{
    auto gtid=blockDim.x*blockIdx.x+threadIdx.x;
    if (gtid<n)
    {
        c[gtid]=a[gtid]+b[gtid];
    }
}
void run_normal_kernel(const int &sz,const int &bytes,dim3 &grid,dim3 &block)
{
    std::vector<int>h_a(sz);
    std::vector<int>h_b(sz);
    std::vector<int>h_c(sz,0);
    init(h_a),init(h_b);
    int *d_a=nullptr,*d_b=nullptr,*d_c=nullptr;
    hipMalloc(reinterpret_cast<void**>(&d_a),bytes);
    hipMalloc(reinterpret_cast<void**>(&d_b),bytes);
    hipMalloc(reinterpret_cast<void**>(&d_c),bytes);
    hipMemcpy(d_a,h_a.data(),bytes,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b.data(),bytes,hipMemcpyHostToDevice);
    kernel<<<grid,block>>>(d_a,d_b,d_c,sz);
    hipDeviceSynchronize();
    hipMemcpy(h_c.data(),d_c,bytes,hipMemcpyDeviceToHost);
    hipFree(d_a),hipFree(d_b),hipFree(d_c);
    h_a.clear(),h_b.clear(),h_c.clear();
}
void run_zero_cpy_memory(const int &sz,const int &bytes,dim3 &grid,dim3 &block)
{
    int *h_a=nullptr;
    int *h_b=nullptr;
    int *h_c=nullptr;
    int *d_a,*d_b,*d_c;
    hipHostAlloc(reinterpret_cast<void**>(&h_a),bytes,hipHostMallocMapped);
    hipHostAlloc(reinterpret_cast<void**>(&h_b),bytes,hipHostMallocMapped);
    hipHostAlloc(reinterpret_cast<void**>(&h_c),bytes,hipHostMallocMapped);
    init(h_a,sz),init(h_b,sz);
    hipHostGetDevicePointer(reinterpret_cast<void**>(&d_a),(void*)h_a,0);
    hipHostGetDevicePointer(reinterpret_cast<void**>(&d_b),(void*)h_b,0);
    hipHostGetDevicePointer(reinterpret_cast<void**>(&d_c),(void*)h_c,0);
    kernel<<<grid,block>>>(d_a,d_b,d_c,sz);
    hipDeviceSynchronize();
    hipHostFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);

}
int main(int argc,const char *argv [])
{
    int dev_idx=0;
    hipDeviceProp_t dev_pro;
    hipGetDeviceProperties(&dev_pro,dev_idx);
    if (!dev_pro.canMapHostMemory)
    {
        std::cout<<"doesn't provide zero CPY"<<std::endl;
        hipDeviceReset();
        exit(EXIT_SUCCESS);
    }
    int sz=1<<22;
    auto bytes=sz*sizeof(sz);
    dim3 block(512);
    dim3 grid((sz+block.x-1)/block.x);
    run_normal_kernel(sz,bytes,grid,block);
    run_zero_cpy_memory(sz,bytes,grid,block);
    return 0;
}