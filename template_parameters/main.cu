#include<bits/stdc++.h>
#include<hip/hip_runtime.h>

void seq_array_accum(int &a, const std::vector<int> &arr)
{
    for (const auto &i : arr)
    {
        a+=i;
    }
}
void init(std::vector<int>&arr)
{
    for (int i=0;i<static_cast<int>(arr.size());++i)
    {
        arr[i]=i%10;
    }
}

template<unsigned int iblock_size>
__global__ void kernel(int *input,int *part,int size)
{
    auto tid=threadIdx.x;
    auto index=tid+blockDim.x*blockIdx.x*8;
    int *window=input+blockDim.x*blockIdx.x*8;
    if ((index+7*blockDim.x)<size)
    {
        int a1=input[index+blockDim.x];
        int a2=input[index+2*blockDim.x];
        int a3=input[index+3*blockDim.x];
        int a4=input[index+4*blockDim.x];
        int a5=input[index+5*blockDim.x];
        int a6=input[index+6*blockDim.x];
        int a7=input[index+7*blockDim.x];
        input[index]+=a1+a2+a3+a4+a5+a6+a7;
    }
    __syncthreads();
    if (iblock_size>=1024&&tid<512)
    {
        window[tid]+=window[tid+512];
    }
    __syncthreads();
    if (iblock_size>=512&&tid<256)
    {
        window[tid]+=window[tid+256];
    }
    __syncthreads();
    if (iblock_size>=256&&tid<128)
    {
        window[tid]+=window[tid+128];
    }
    __syncthreads();
    if (iblock_size>=128&&tid<64)
    {
        window[tid]+=window[tid+64];
    }
    __syncthreads();
    if (tid<32)
    {
        volatile int *vsmem=window;
        vsmem[tid]+=vsmem[tid+32];
        vsmem[tid]+=vsmem[tid+16];
        vsmem[tid]+=vsmem[tid+8];
        vsmem[tid]+=vsmem[tid+4];
        vsmem[tid]+=vsmem[tid+2];
        vsmem[tid]+=vsmem[tid+1];
    }
    __syncthreads();
    if (tid==0)
    {
        part[blockIdx.x]=window[0];
    }
    __syncthreads();

}
int main()
{
    constexpr auto size=1<<27;
    constexpr auto block_size=1024;
    auto cpu_res=0,gpu_res=0;
    dim3 blocks(block_size);
    dim3 grid((size/block_size)/8);
    constexpr auto input_byte_size=size*sizeof(int);
    const auto part_byte_size=grid.x*sizeof(int);
    std::vector<int>h_input(size);
    std::vector<int>h_part(grid.x,0);
    int *d_input=nullptr,*d_part=nullptr;
    init(h_input);
    seq_array_accum(cpu_res,h_input);
    hipMalloc(reinterpret_cast<void**>(&d_input),input_byte_size);
    hipMalloc(reinterpret_cast<void**>(&d_part),part_byte_size);
    hipMemset(d_part,0,part_byte_size);
    hipMemcpy(d_input,h_input.data(),input_byte_size,hipMemcpyHostToDevice);
    switch (block_size)
    {
    case 1024:
        kernel<1024><<<grid,blocks>>>(d_input,d_part,size);
        break;
    case 512:
        kernel<512><<<grid,blocks>>>(d_input,d_part,size);
        break;
    case 256:
        kernel<256><<<grid,blocks>>>(d_input,d_part,size);
        break;
    case 128:
        kernel<128><<<grid,blocks>>>(d_input,d_part,size);
        break;
    case 64:
        kernel<64><<<grid,blocks>>>(d_input,d_part,size);
        break;
    }
    hipDeviceSynchronize();
    hipMemcpy(h_part.data(),d_part,part_byte_size,hipMemcpyDeviceToHost);
    seq_array_accum(gpu_res,h_part);
    std::cout<<gpu_res<<" "<<cpu_res<<"\n";
    std::cout<<(gpu_res==cpu_res)<<"\n";
    return 0;
}