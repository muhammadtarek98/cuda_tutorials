#include<bits/stdc++.h>
#include <hip/hip_runtime.h>

__global__ void kernel(int size,int depth)
{
    printf("depth: %d - thread index:%d \n",depth,threadIdx.x);
    if (size==1){return;}
    if (threadIdx.x==0)
    {
        kernel<<<1,size/2>>>(size/2,depth+1);
    }
}
int main(int argc,const char *argv[])
{
    kernel<<<1,16>>>(16,0);
    hipDeviceSynchronize();
    hipDeviceReset();
    /**/
    return 0;
}