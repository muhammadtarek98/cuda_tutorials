#include <bits/stdc++.h>
#include<hip/hip_runtime.h>

__global__ void kernel(int *a)
{
    int tidx=blockDim.x*threadIdx.y+threadIdx.x;
    int threads_per_block=blockDim.x*blockDim.y;
    int block_offset=blockIdx.x*threads_per_block;
    int threads_per_row=threads_per_block*gridDim.x;
    int row_offset=threads_per_row*blockIdx.y;
    int tidy=row_offset+block_offset+tidx;
    printf("blockidx.x: %d blockidx.y: %d threadidx.x:%d threadid.y:%d value:%d\n",blockIdx.x ,blockIdx.y,
        tidx ,tidy,a[tidy]);
}
int main()
{
    dim3 block (2,2),grid(2,2);
    int n=16,nbytes=sizeof(int)*n;
    int h_arr[]{1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16};
    int *d_arr;
    hipMalloc((void**)&d_arr,nbytes);
    hipMemcpy(d_arr,h_arr,nbytes,hipMemcpyHostToDevice);
    kernel<<<grid,block>>>(d_arr);
    hipDeviceSynchronize();
    hipDeviceReset();

    return 0;
}