#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

const int nx=1024,ny=1024,block_x=128,block_y=8;

void print_matrix(std::vector<int> &matrix)
{
    for (int i = 0; i < nx; ++i)
    {
        for (int j = 0; j < ny; ++j)
        {
            printf("%d ",matrix[i+j]);
        }
    }
    printf("\n");
}
void init_matrix(std::vector<int> &a)
{
    int j=0;
    for (int i = 0; i < a.size(); ++i)
    {
        if (i % 5 == 0)
        {
            a[i] = i+j;
        }
        else
        {
            a[i]= i*j+7;
        }
    }
}
__global__ void row_major_kernel(int *a,int *a_trans,int nx,int ny)
{
    int ix=threadIdx.x+blockDim.x*blockIdx.x;
    int iy=threadIdx.y+blockDim.y*blockIdx.y;
    if (ix<nx&&iy<ny)
    {
        a_trans[ix*ny+iy]=a[iy*nx+ix];
    }
    __syncthreads();
}
__global__ void col_major_kernel(int *a,int *a_trans,int nx,int ny)
{
    int ix=threadIdx.x+blockDim.x*blockIdx.x;
    int iy=threadIdx.y+blockDim.y*blockIdx.y;
    if (ix<nx&&iy<ny)
    {
        a_trans[iy*nx+ix]=a[ix*ny+iy];
    }
    __syncthreads();
}
__global__ void row_major_unrolling(int *a,int *trans,int nx,int ny)
{
    int ix=threadIdx.x+blockDim.x*blockIdx.x*4;
    int iy=threadIdx.y+blockDim.y*blockIdx.y;
    int ti=iy*nx+ix,to=ix*ny+iy;

    if (ix+3*blockDim.x<nx&&iy<ny)
    {
        trans[to]=a[ti];
        trans[to+blockDim.x*ny]=a[ti+blockDim.x];
        trans[to+2*blockDim.x*ny]=a[ti+2*blockDim.x];
        trans[to+3*blockDim.x*ny]=a[ti+3*blockDim.x];
    }
    __syncthreads();
}
__global__ void col_major_unrolling(int *a,int *trans,int nx,int ny)
{
    int ix=threadIdx.x+blockDim.x*blockIdx.x*4;
    int iy=threadIdx.y+blockDim.y*blockIdx.y;
    int ti=iy*nx+ix,to=ix*ny+iy;
    if (ix+3*blockDim.x<nx&&iy<ny)
    {
        trans[ti]=a[to];
        trans[ti+blockDim.x]=a[to+blockDim.x*ny];
        trans[ti+2*blockDim.x]=a[to+2*blockDim.x*ny];
        trans[ti+3*blockDim.x]=a[to+3*blockDim.x*ny];
    }
    __syncthreads();

}
__global__ void diagonal_major(int *a,int *trans,int nx,int ny)
{
    int blk_x=blockIdx.x,blk_y=(blockIdx.x+blockIdx.y)%gridDim.x;
    int ix=blockIdx.x*blk_x+threadIdx.x;
    int iy=blockIdx.y*blk_y+threadIdx.y;
    if (ix<nx&&iy<ny)
    {
        trans[ix*ny+iy]=a[iy*nx+ix];
    }
    __syncthreads();
}
void run_diagonal_major_kernel()
{
    printf("run_diagonal_major_kernel\n");
    int sz=ny*nx;
    int bytes=sizeof(int)*sz;
    dim3 blocks(block_x,block_y);
    dim3 grid((nx + block_x - 1) / block_x,(ny + block_y - 1) / block_y);
    std::vector<int> h_a(nx*ny,0);
    std::vector<int>h_a_trans(ny*nx,0);
    int *d_a=nullptr,*d_a_trans=nullptr;
    hipMalloc(reinterpret_cast<void**>(&d_a),bytes);
    hipMalloc(reinterpret_cast<void**>(&d_a_trans),bytes);
    init_matrix(h_a);
    hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
    row_major_kernel<<<grid,blocks>>>(d_a,d_a_trans,nx,ny);
    hipDeviceSynchronize();
    hipMemcpy(h_a_trans.data(),d_a_trans,bytes,hipMemcpyDeviceToHost);
    hipFree(d_a),hipFree(d_a_trans);
    print_matrix(h_a_trans);
    h_a.clear(),h_a_trans.clear();
    hipDeviceReset();
}
void run_row_major_kernel()
{
    printf("run_row_major_kernel\n");
    int sz=ny*nx;
    int bytes=sizeof(int)*sz;
    dim3 blocks(block_x,block_y);
    dim3 grid((nx + block_x - 1) / block_x,(ny + block_y - 1) / block_y);
    std::vector<int> h_a(nx*ny,0);
    std::vector<int>h_a_trans(ny*nx,0);
    int *d_a=nullptr,*d_a_trans=nullptr;
    hipMalloc(reinterpret_cast<void**>(&d_a),bytes);
    hipMalloc(reinterpret_cast<void**>(&d_a_trans),bytes);
    init_matrix(h_a);
    hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
    row_major_kernel<<<grid,blocks>>>(d_a,d_a_trans,nx,ny);
    hipDeviceSynchronize();
    hipMemcpy(h_a_trans.data(),d_a_trans,bytes,hipMemcpyDeviceToHost);
    hipFree(d_a),hipFree(d_a_trans);
    print_matrix(h_a_trans);
    h_a.clear(),h_a_trans.clear();
    hipDeviceReset();
}
void run_col_major_kernel()
{
    printf("run_col_major_kernel\n");
    int sz=ny*nx;
    int bytes=sizeof(int)*sz;
    dim3 blocks(block_x,block_y);
    dim3 grid((nx + block_x - 1) / block_x,(ny + block_y - 1) / block_y);;
    std::vector<int> h_a(nx*ny,0);
    std::vector<int>h_a_trans(ny*nx,0);
    int *d_a=nullptr,*d_a_trans=nullptr;
    init_matrix(h_a);
    hipMalloc(reinterpret_cast<void**>(&d_a),bytes);
    hipMalloc(reinterpret_cast<void**>(&d_a_trans),bytes);
    hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
    col_major_kernel<<<grid,blocks>>>(d_a,d_a_trans,nx,ny);
    hipDeviceSynchronize();
    hipMemcpy(h_a_trans.data(),d_a_trans,bytes,hipMemcpyDeviceToHost);
    hipFree(d_a),hipFree(d_a_trans);
    print_matrix(h_a_trans);
    h_a.clear(),h_a_trans.clear();
    hipDeviceReset();
}

void run_col_major_unroll_kernel()
{
    printf("run_col_major_unroll_kernel\n");
    int sz=ny*nx;
    int bytes=sizeof(int)*sz;
    dim3 blocks(block_x,block_y);
    dim3 grid((nx + block_x - 1) / block_x,(ny + block_y - 1) / block_y);;
    std::vector<int> h_a(nx*ny);
    std::vector<int>h_a_trans(ny*nx);
    int *d_a=nullptr,*d_a_trans=nullptr;
    hipMalloc(reinterpret_cast<void**>(&d_a),bytes);
    hipMalloc(reinterpret_cast<void**>(&d_a_trans),bytes);
    init_matrix(h_a);
    hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
    col_major_unrolling<<<grid,blocks>>>(d_a,d_a_trans,nx,ny);
    hipDeviceSynchronize();
    hipMemcpy(h_a_trans.data(),d_a_trans,bytes,hipMemcpyDeviceToHost);
    hipFree(d_a),hipFree(d_a_trans);
    print_matrix(h_a_trans);
    h_a.clear(),h_a_trans.clear();
    hipDeviceReset();
}
void run_row_major_unroll_kernel()
{
    printf("run_row_major_unroll_kernel\n");
    int sz=ny*nx;
    int bytes=sizeof(int)*sz;
    dim3 blocks(block_x,block_y);
    dim3 grid((nx + block_x - 1) / block_x,(ny + block_y - 1) / block_y);;
    std::vector<int> h_a(nx*ny);
    std::vector<int>h_a_trans(nx*ny);
    init_matrix(h_a);
    int *d_a=nullptr,*d_a_trans=nullptr;
    hipMalloc(reinterpret_cast<void**>(&d_a),bytes);
    hipMalloc(reinterpret_cast<void**>(&d_a_trans),bytes);
    hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
    row_major_unrolling<<<grid,blocks>>>(d_a,d_a_trans,nx,ny);
    hipDeviceSynchronize();
    hipMemcpy(h_a_trans.data(),d_a_trans,bytes,hipMemcpyDeviceToHost);
    hipFree(d_a),hipFree(d_a_trans);
    print_matrix(h_a_trans);
    h_a.clear(),h_a_trans.clear();
    hipDeviceReset();
}

int main()
{
    run_row_major_kernel();
    run_col_major_kernel();
    run_col_major_unroll_kernel();
    run_row_major_unroll_kernel();
    run_diagonal_major_kernel();
    return 0;
}