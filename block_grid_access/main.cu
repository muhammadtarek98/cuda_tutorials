#include <iostream>
#include <hip/hip_runtime.h>

__global__ void print_details()
{
    printf("threadIdx.x: %d threadIdx.y: %d threadIdx.z: %d blockIdx.x: %d blockIdx.y: %d blockIdx.z: %d blockDim.x: %d blockDim.y: %d blockDim.z: %d gridDim.x: %d gridDim.y: %d gridDim.z: %d\n"
        ,threadIdx.x,threadIdx.y,threadIdx.z,blockIdx.x,blockIdx.y,blockIdx.z,blockDim.x,blockDim.y,blockDim.z,gridDim.x,gridDim.y,gridDim.z);
}
int main()
{
    int nx=16,ny=16;
    dim3 block(nx/2,ny/2);
    dim3 grid(nx/block.x,ny/block.y);
    print_details<<<grid,block>>>();
    hipDeviceSynchronize();
    hipDeviceReset();

    return 0;
}