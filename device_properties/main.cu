#include <bits/stdc++.h>
#include<hip/hip_runtime.h>

int main()
{
    int device_counter=0;
    hipGetDeviceCount(&device_counter);
    if (device_counter==0)
    {
        std::cout<<"No cuda Device support"<<std::endl;
    }
    int device_idx=0;
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props,device_idx);
    std::cout<<  props.multiProcessorCount<<std::endl;
    std::cout<<  props.maxThreadsPerBlock<<std::endl;
    std::cout<< props.totalGlobalMem/1024.0<<std::endl ;
    std::cout<< props.sharedMemPerBlock/1024.0<<std::endl;
    std::cout<< props.maxThreadsPerMultiProcessor<<std::endl;
    std::cout<<props.maxGridSize<<std::endl;
    std::cout<< props.maxBlocksPerMultiProcessor<<std::endl;

    return 0;
}