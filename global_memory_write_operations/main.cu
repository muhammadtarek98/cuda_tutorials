
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
__global__ void kernel(float *a,float *b,float *c,int size,int offset){
int tgid=threadIdx.x+blockDim.x*blockIdx.x;
    int k=tgid+offset;
    if (k<size)
    {
        c[k]=a[tgid]+b[tgid];
    }
}
template<typename T>
void init(std::vector<T>&arr)
{
    for (int i=0;i<(int)arr.size();++i)
    {
        arr[i]=i%10;
    }
}

int main()
{
    int size=1<<25,block_size=128,offset=5;
    auto bytes=sizeof(float)*size;
    dim3 blocks(block_size),grid((size+block_size-1)/block_size);
    std::vector<float> a(size),b(size),c(size,0.0);
    init<float>(a),init<float>(b);
    float *da=nullptr,*db=nullptr,*dc=nullptr;
    hipMalloc(reinterpret_cast<void**>(&da),bytes);
    hipMemcpy(da,a.data(),bytes,hipMemcpyHostToDevice);

    hipMalloc(reinterpret_cast<void**>(&da),bytes);
    hipMemcpy(db,b.data(),bytes,hipMemcpyHostToDevice);
    hipMalloc(reinterpret_cast<void**>(&dc),bytes);
    hipMemset(dc,0.0,bytes);
    kernel<<<grid,blocks>>>(da,db,dc,size,offset);
    hipDeviceSynchronize();
    hipMemcpy(c.data(),dc,bytes,hipMemcpyDeviceToHost);
    hipFree(da),hipFree(db),hipFree(dc);
    return 0;
}