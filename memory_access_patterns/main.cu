#include <hip/hip_runtime.h>

#include <bits/stdc++.h>
__global__ void kernel(float *a,float *b,float *c,int size,int offset)
{
    int tgid=threadIdx.x+blockDim.x*blockIdx.x;
    int window=tgid+offset;
    if (window<size)
    {
        c[tgid]=a[window]+b[window];
    }
    __syncthreads();
}
template<typename  T>
void init(std::vector<T>&arr)
{
    for (int i=0;i<(int)arr.size();++i)
    {
        arr[i]=i%10;
    }
}
int main()
{
    /*
     * to enable l1 cache while compiling by:
        nvcc -xptxa -dlcm=ca -o -main.out main.cu
    * to disable l1 cache while compiling by:
        nvcc -xptxa -dlcm=cg -o -main.out main.cu
    then profile that with nvfpro
     */
    int size=1<<25,offset=9,block_size=128;
    dim3 blocks(block_size),grids((size+block_size-1)/block_size);
    auto bytes=sizeof(float)*size;
    std::vector<float>a(size),b(size),c(size,0.0);
    init<float>(a),init<float>(b);
    float *d_a=nullptr,*d_b=nullptr,*d_c=nullptr;
    hipMalloc(reinterpret_cast<void**>(&d_a),bytes);
    hipMemcpy(a.data(),d_a,bytes,hipMemcpyHostToDevice);

    hipMalloc(reinterpret_cast<void**>(&d_b),bytes);
    hipMemcpy(b.data(),d_b,bytes,hipMemcpyHostToDevice);
    hipMalloc(reinterpret_cast<void**>(&d_c),bytes);
    hipMemset(reinterpret_cast<void**>(&d_c),0.0,bytes);
    kernel<<<grids,blocks>>>(d_a,d_b,d_c,size,offset);
    hipDeviceSynchronize();
    hipMemcpy(d_c,c.data(),bytes,hipMemcpyDeviceToHost);
    hipFree(d_c),hipFree(d_a),hipFree(d_b);

    return 0;
}