#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#define gpuErrrchk(res){gpu_raise((res),__FILE__,__LINE__);}
inline void gpu_raise(hipError_t code,const char *file,int line,bool terminate=true)
{
    if (code!=hipSuccess)
    {
        std::cerr<<stderr<<"GPU error:\n"<<hipGetErrorString(code)<<file <<line<<"\n";
        if (terminate)
        {
            exit(code);
        }
    }
}
extern "C"{
    __global__ void kernel(int *a,int *b,int *c,int sz)
    {
        int gid=blockIdx.x*blockDim.x+threadIdx.x;
        if (gid<sz){
            c[gid]+=a[gid]+b[gid];
            printf("value:%d, threadidx:%d, blockidx:%d \n",c[gid],threadIdx.x,blockIdx.x);
        }
    }
}
void random_ints(int* arr, int size) {
    srand(time(NULL));

    for(int i = 0; i < size; i++) {
        arr[i] = rand();
    }
}
int main()
{
    int sz=128,n_blocks=2;
    size_t nbytes=sz*sizeof(int);
    clock_t gpu_start,gpu_end;
    hipError_t cuda_error;
    dim3 blocks(sz/n_blocks);
    dim3 grids(n_blocks);
    int *h_a,*h_b,*h_c,*d_a,*d_b,*d_c;
    h_a=(int*)malloc(nbytes);
    h_b=(int*)malloc(nbytes);
    h_c=(int*)malloc(nbytes);
    memset(h_c,0,nbytes);
    random_ints(h_a,sz);
    random_ints(h_b,sz);
    gpuErrrchk(hipMalloc((void**)&d_a,nbytes));
    gpuErrrchk(hipMalloc((void**)&d_b,nbytes));
    gpuErrrchk(hipMalloc((void**)&d_c,nbytes));
    gpuErrrchk(hipMemcpy(d_a,h_a,nbytes,hipMemcpyHostToDevice));
    gpuErrrchk(hipMemcpy(d_b,h_b,nbytes,hipMemcpyHostToDevice));
    gpu_start=clock();
    kernel<<<grids,blocks>>>(d_a,d_b,d_c,sz);
    hipDeviceSynchronize();
    gpuErrrchk(hipMemcpy(h_c,d_c,nbytes,hipMemcpyDeviceToHost));
    gpu_end=clock();
    std::cout<<(double)(gpu_end-gpu_start)/CLOCKS_PER_SEC<<" seconds"<<std::endl;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipDeviceReset();
    delete h_a;
    delete h_b;
    delete h_c;
    return 0;
}