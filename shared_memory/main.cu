#include <bits/stdc++.h>
#include<hip/hip_runtime.h>

#define SHARED_MEM_SIZE 128
const int sz=1<<22,block_size=SHARED_MEM_SIZE,bytes=sizeof(int)*sz;
dim3 blocks(block_size);
dim3 grids((sz/blocks.x)+1);
__global__ void kernel_1(int *a,int *b)
{
    int tid=threadIdx.x;
    int tgid=blockDim.x*blockIdx.x+tid;
    __shared__ int s_arr[SHARED_MEM_SIZE];
    if (tgid<sz)
    {
        s_arr[tid]=a[tgid];
        b[tgid]=s_arr[tid];
    }
}
__global__ void kernel_2(int *a,int *b)
{
    int tid=threadIdx.x;
    int tgid=tid+blockDim.x*blockIdx.x;
    extern __shared__ int s_arr[];
    if (tgid<sz)
    {
        s_arr[tid]=a[tgid];
        b[tgid]=s_arr[tid];
    }
}
void init(std::vector<int>&arr)
{
    for (int i=0;i<(int)arr.size();++i)
    {
        arr[i]=i%10;
    }
}
void run_static_shared_mem()
{
    std::vector<int>h_a(sz),h_b(sz);
    int *d_a=nullptr,*d_b=nullptr;
    hipMalloc(reinterpret_cast<void**>(&d_a),bytes);
    hipMalloc(reinterpret_cast<void**>(&d_b),bytes);
    init(h_a);
    hipMemcpy(d_a,h_a.data(),bytes,hipMemcpyHostToDevice);
    kernel_1<<<grids,blocks>>>(d_a,d_b);
    hipDeviceSynchronize();
    hipMemcpy(h_b.data(),d_b,bytes,hipMemcpyDeviceToHost);
    hipFree(d_a),hipFree(d_b);
    h_a.clear(),h_b.clear();
}
void run_dynamic_shared_mem()
{
    std::vector<int>h_a(sz),h_b(sz);
    int *d_a=nullptr,*d_b=nullptr;
    hipMalloc(reinterpret_cast<void**>(&d_a),bytes);
    hipMalloc(reinterpret_cast<void**>(&d_b),bytes);
    init(h_a);
    hipMemcpy(d_a,h_a.data(),bytes,hipMemcpyHostToDevice);
    kernel_2<<<grids,blocks,SHARED_MEM_SIZE*sizeof(int)>>>(d_a,d_b);
    hipDeviceSynchronize();
    hipMemcpy(h_b.data(),d_b,bytes,hipMemcpyDeviceToHost);
    hipFree(d_a),hipFree(d_b);
    h_a.clear(),h_b.clear();
}
int main()
{
    run_static_shared_mem();
    run_dynamic_shared_mem();
    return 0;
}