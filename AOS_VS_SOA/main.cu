#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

#define  N 1<<22
#define BS 128
struct AOS
{
    float x, y;
    AOS():x(0.0),y(0.0){}
};
struct SOA
{
    float x[N],y[N];
};
__global__ void aos_kernel(AOS *in,AOS *out,const int size){
    auto tgid=threadIdx.x+blockDim.x*blockIdx.x;
    if (tgid<size)
    {
        AOS temp=in[tgid];
        temp.x+=5,temp.y+=10;
        out[tgid]=temp;
        printf("AOS :threadIdx:%d in_element.x:%f in_element.y:%f out_element.x:%f out_element.y:%f\n",
        tgid,in[tgid].x,in[tgid].y,out[tgid].x,out[tgid].y);
    }
    __syncthreads();
}
__global__ void soa_kernel(SOA *in,SOA *out,const int size)
{
    auto tgid=threadIdx.x+blockDim.x*blockIdx.x;
    if (tgid<size)
    {
        float tmpx = in->x[static_cast<int>(tgid)];
        float tmpy = in->y[static_cast<int>(tgid)];
        tmpx += 5;
        tmpy += 10;
        out->x[tgid] = tmpx;
        out->y[tgid] = tmpy;
        printf("SOA: threadIdx:%d in_element.x:%f in_element.y:%f out_element.x:%f out_element.y:%f\n",
            tgid,in->x[tgid],in->y[tgid],out->x[tgid],out->y[tgid]);
    }
    __syncthreads();
}
void run_AOS()
{
    auto bytes=sizeof(AOS)*N;
    dim3 blocks(BS);
    dim3 grid(N/BS);
    std::vector<AOS>in(N);
    std::vector<AOS>out(N);
    std::shared_ptr<AOS>d_in,d_out;
    hipMalloc(reinterpret_cast<void**>(&d_in),bytes);
    hipMalloc(reinterpret_cast<void**>(&d_out),bytes);
    hipMemcpy(d_in.get(),in.data(),bytes,hipMemcpyHostToDevice);
    aos_kernel<<<grid,blocks>>>(d_in.get(),d_out.get(),N);
    hipDeviceSynchronize();
    hipMemcpy(out.data(),d_out.get(),bytes,hipMemcpyDeviceToHost);
    hipFree(d_in.get()),hipFree(d_out.get());
    hipDeviceReset();
    in.clear(),out.clear();
}
void run_SOA()
{
    auto bytes=sizeof(SOA);
    dim3 blocks(BS),grid(N/BS);
    std::shared_ptr<SOA>in,out;
    std::shared_ptr<SOA>d_in,d_out;
    hipMalloc(reinterpret_cast<void**>(&d_in),bytes);
    hipMalloc(reinterpret_cast<void**>(&d_out),bytes);
    hipMemcpy(in.get(),d_in.get(),bytes,hipMemcpyHostToDevice);
    soa_kernel<<<grid,blocks>>>(d_in.get(),d_out.get(),N);
    hipDeviceSynchronize();
    hipMemcpy(d_out.get(),out.get(),bytes,hipMemcpyDeviceToHost);
    hipFree(d_in.get()),hipFree(d_out.get());
    in.reset();
    out.reset();
    hipDeviceReset();
}
int main()
{
    run_AOS();
    run_SOA();
    return 0;
}