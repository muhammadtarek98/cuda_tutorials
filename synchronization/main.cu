#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

bool compare_results(const int &cpu_res,const int &gpu_res)
{
        if (gpu_res!=cpu_res)
        {
            return false;
        }
    return true;
}
void init(std::vector<int>&arr)
{
    for (int i=0;i<(int)arr.size();++i)
    {
        arr[i]=i%10;
    }
}
__global__ void arr_accum(int *arr,int *chunk,int size)
{
    auto tid=threadIdx.x;
    auto tgid=blockDim.x*blockIdx.x+tid;
    if (tgid>size)
    {
        return;
    }
    for (int offset=1;offset<=blockDim.x/2;offset*=2)
    {
         if (tid%(2*offset)==0)
         {
             arr[tgid]+=arr[tgid+offset];
         }
        __syncthreads();
    }
    if (tid==0)
    {
        chunk[blockIdx.x]=arr[tgid];
    }

}
void seq_array_accum(int &a, const std::vector<int> &arr)
{
    for (const auto &i : arr)
    {
        a+=i;
    }
}

int main(){
    constexpr int size=1<<27;
    constexpr int block_size=128;
    int cpu_res=0,gpu_res=0;
    dim3 block(block_size);
    dim3 grid(size/block.x);
    constexpr auto input_byte_size=size*sizeof(int);
    const auto part_byte_size=grid.x*sizeof(int);
    std::vector<int> h_input(size);
    std::vector<int>h_part(grid.x);
    int *d_input=nullptr;
    int *d_part=nullptr;
    init(h_input);
    seq_array_accum(cpu_res,h_input);
    hipMalloc(reinterpret_cast<void**>(&d_input),input_byte_size);
    hipMalloc(reinterpret_cast<void**>(&d_part),part_byte_size);
    hipMemset(d_part,0,part_byte_size);
    hipMemcpy(d_input,h_input.data(),input_byte_size,hipMemcpyHostToDevice);
    arr_accum<<<grid,block>>>(d_input,d_part,size);
    hipDeviceSynchronize();
    hipMemcpy(h_part.data(),d_part,part_byte_size,hipMemcpyDeviceToHost);
    seq_array_accum(gpu_res,h_part);
    std::cout<<compare_results(cpu_res,gpu_res)<<std::endl;
    std::cout<<cpu_res<<" "<<gpu_res;
    hipFree(d_input);
    hipFree(d_part);
    h_input.clear();
    h_part.clear();
    hipDeviceReset();
    return 0;
}