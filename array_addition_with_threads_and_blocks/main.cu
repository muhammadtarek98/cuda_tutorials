
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
void random_ints(int* arr, int size) {
    srand(time(NULL));
    for(int i = 0; i < size; i++) {
        arr[i] = int(rand() % 100);
    }
}

__global__ void add(int *a,int *b,int *c)
{
    int idx=threadIdx.x+blockIdx.x*blockDim.x;
    c[idx]=a[idx]+b[idx];
}
void print(int* arr,int size)
{
    for (int i=0;i<size;i++)
    {
        std::cout<<arr[i]<<" ";
    }
}
int main()
{
    int N=2048*2048;
    int n_thread_per_block=512;
    int size=N*sizeof(int);
    int *a=nullptr, *b=nullptr, *c=nullptr;
    int *d_a=nullptr,*d_b=nullptr,*d_c=nullptr;
    hipMalloc((void **)&d_a,size);
    hipMalloc((void **)&d_b,size);
    hipMalloc((void **)&d_c,size);
    a=(int *)malloc(size);random_ints(a,N);
    b=(int *)malloc(size);random_ints(b,N);
    c=(int *)malloc(size);
    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
    add<<<N/n_thread_per_block,n_thread_per_block>>>(d_a,d_b,d_c);
    hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
    print(c,N);
    free(a),free(b),free(c);
    hipFree(d_a),hipFree(d_b),hipFree(d_c);

    return 0;
}