#include <bits/stdc++.h>
#include "hip/hip_runtime.h"


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}
enum INIT_PARAM{
	INIT_ZERO,INIT_RANDOM,INIT_ONE,INIT_ONE_TO_TEN,INIT_FOR_SPARSE_METRICS,INIT_0_TO_X
};


void print_array(int * input, const int array_size)
{
	for (int i = 0; i < array_size; i++)
	{
		if (!(i == (array_size - 1)))
		{
			printf("%d,", input[i]);
		}
		else
		{
			printf("%d \n", input[i]);
		}
	}
}
//simple initialization
void init(std::vector<int>&arr)
{
	for (int i=0;i<(int)arr.size();++i)
	{
		arr[i]=i%10;
	}
}
void seq_array_accum(int &a, const std::vector<int> &arr)
{
	for (const auto &i : arr)
	{
		a+=i;
	}
}

bool compare_results(const int &cpu_res,const int &gpu_res)
{
	if (gpu_res!=cpu_res)
	{
		return false;
	}
	std::cout<<cpu_res<<"="<<gpu_res<<std::endl;
	return true;
}
void compare_arrays(float * a, float * b, float size)
{
	for (int i = 0; i < size; i++)
	{
		if (a[i] != b[i])
		{
			printf("Arrays are different \n");

			return;
		}
	}
	printf("Arrays are same \n");

}
__global__ void interleaved_pair(int *input,int *part,const int size)
{
	const auto tid=threadIdx.x;
	const auto tgid=tid+blockDim.x*blockIdx.x;
	if (tgid>size){return;}
	for (auto offset=blockDim.x/2;offset>0;offset/=2)
	{
		if (tid<offset)
		{
			input[tgid]+=input[tgid+offset];
		}
		__syncthreads();
	}
	if (tid==0)
	{
		part[blockIdx.x]=input[tgid];
	}

}

__global__ void reduction_neighbored_pairs_improved(int *input,int *part,const int size)
{
	const auto tid = threadIdx.x;
	const auto tgid = blockDim.x * blockIdx.x + tid;
	//local window data
	int *window = input + blockDim.x * blockIdx.x;
	if (tgid > size)
		return;
	for (int offset = 1; offset <= blockDim.x /2 ; offset *= 2)
	{
		auto index = 2 * offset * tid;
		if (index < blockDim.x)
		{
			window[index] += window[index + offset];
		}
		__syncthreads();
	}
	if (tid == 0)
	{
		part[blockIdx.x] = input[tgid];
	}
}

int main()
{
	int size = 1 << 27,block_size = 128,cpu_res=0,gpu_res=0;
	dim3 block(block_size);
	dim3 grid(size / block.x);
	const auto input_byte_size = size * sizeof(int);
	const auto part_byte_size = grid.x*sizeof(int);
	std::vector<int> h_input(size);
	std::vector<int> h_part(grid.x);
 	init(h_input);
	int *d_input=nullptr;
	int *d_part=nullptr;
	hipMalloc(reinterpret_cast<void**>(&d_input),input_byte_size);
	hipMalloc(reinterpret_cast<void**>(&d_part),part_byte_size);
	hipMemset(d_part,0,part_byte_size);
	seq_array_accum(cpu_res,h_input);
	hipMemcpy(d_input,h_input.data(),input_byte_size,hipMemcpyHostToDevice);
	interleaved_pair<<<grid,block>>>(d_input,d_part,size);
	hipDeviceSynchronize();
	hipMemcpy(h_part.data(),d_part,part_byte_size,hipMemcpyDeviceToHost);
	seq_array_accum(gpu_res,h_part);
	std::cout<<compare_results(cpu_res,gpu_res)<<std::endl;
	hipFree(d_input),hipFree(d_part);


	return 0;
}