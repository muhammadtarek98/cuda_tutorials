
#include <hip/hip_runtime.h>
#include <iostream>
__global__ void add(const int *a,const int *b,int *c)
{
    *c=*a+*b;
}
int main()
{
    //create host data
    int a=7,b=8,c=0;
    //create device pointers for host data
    int *d_a=&a,*d_b=&b,*d_c=&c;
    //get the size of the host data
    int size=sizeof(int);
    hipMalloc((void**)&d_a,size);
    //allocate device memory
    hipMalloc((void**)&d_b,size);
    hipMalloc((void**)&d_c,size);
    //copy the data from host to device
    hipMemcpy(d_a,&a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,&b,size,hipMemcpyHostToDevice);
    //execute the kernel function
    add<<<1,1>>>(d_a,d_b,d_c);
    //copy the data from device to host
    hipMemcpy(&c,d_c,size,hipMemcpyDeviceToHost);
    std::cout<<c<<std::endl;
    //deallocate the device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);


    return 0;
}