
#include <hip/hip_runtime.h>
#include <iostream>
__global__ void kernel_1()
{
    int gid=threadIdx.x+blockIdx.x*blockDim.x;
    int warp_id=gid/32;
    float a=0.0,b=0.0;
    if (warp_id%2==0)
    {
        a=100.0,b=50.0;
    }
    else
    {
        a=200.0,b=500.0;
    }
}
__global__ void kernel_2()
{
    int gid=threadIdx.x+blockIdx.x*blockDim.x;
    float a=0.0,b=0.0;
    if (gid%2==0)
    {
        a=100.0,b=50.0;
    }
    else
    {
        a=200.0,b=500.0;
    }
}
int main()
{
    int size=1<<22;
    dim3 block_size(128);
    dim3 grid_size((size+block_size.x-1)/block_size.x);
    kernel_1<<<grid_size,block_size>>>();
    hipDeviceSynchronize();
    kernel_2<<<grid_size,block_size>>>();
    hipDeviceSynchronize();

    return 0;
}