#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

extern "C"{
__global__ void kernel()
{
    int thread_global_id=blockIdx.y*gridDim.x*blockDim.y+blockIdx.x*blockDim.x+threadIdx.x;
    int warp_id=threadIdx.x/32;
    int block_global_id=blockIdx.y*gridDim.x+blockIdx.x;
    printf("thread id:%d \tblock id.x:%d \tblock id.y:%d \tthread global id:%d \twarp id:%d \tblock global id:%d\n",
        threadIdx.x,blockIdx.x,blockIdx.y,thread_global_id,warp_id,block_global_id);
}
}
int main()
{
    dim3 block_size(42);
    dim3 grid_size(2,2);
    kernel<<<grid_size,block_size>>>();
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}