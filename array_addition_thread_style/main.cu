
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
void random_ints(int* arr, int size) {
    srand(time(NULL));

    for(int i = 0; i < size; i++) {
        arr[i] = rand();
    }
}
__global__ void add(int *a,int *b,int *c)
{
    c[threadIdx.x]=a[threadIdx.x]+b[threadIdx.x];
}
int main()
{
    int N=512;
    int size=sizeof(int)*N;
    int *a=(int*)malloc(size);
    int*b=(int*)malloc(size);
    int*c=(int*)malloc(size);
    int*d_a,*d_b,*d_c;
    random_ints(a,N);
    random_ints(b,N);
    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&d_b,size);
    hipMalloc((void**)&d_c,size);
    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
    add<<<1,N>>>(d_a,d_b,d_c);
    hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
    hipFree(d_a),hipFree(d_b),hipFree(d_c);
    for (int i=0;i<size;++i)
    {
        std::cout<<c[i]<<"\n";
    }
    free(a);free(b);free(c);

    return 0;
}