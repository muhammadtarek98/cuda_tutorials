#include "hip/hip_runtime.h"
#include<bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <>
__global__ void kernel(int *input,int *part,unsigned int size)
{
    auto tid=threadIdx.x;
    int *window=input+blockDim.x*blockIdx.x;
    int *outpart=&part[blockIdx.x];
    if (size==2 && tid==0)
    {
        part[blockIdx.x]=input[0]+input[1];
        return;
    }
    __syncthreads();
    int istride=size>>1;
    if (istride>1 && tid<istride)
    {
        window[tid]+=window[tid+istride];
    }
    __syncthreads();
    if (tid==0)
    {
        kernel<<<1,istride>>>(window,outpart,istride);
        hipDeviceSynchronize();
    }
    __syncthreads();

}
void init(std::vector<int>&arr)
{
    for (int i=0;i<static_cast<int>(arr.size());++i)
    {
        arr[i]=i%10;
    }
}
void seq_array_accum(int &a, const std::vector<int> &arr)
{
    for (const auto &i : arr)
    {
        a+=i;
    }
}
int main(int argc,const char *argv[])
{
    int size=1<<22,res=0;
    int block_size=512;
    dim3 blocks(block_size);
    dim3 grid((size/block_size));
    const auto input_byte_size=size*sizeof(int);
    const auto part_byte_size=grid.x*sizeof(int);
    std::vector<int> h_input(size);
    std::vector<int>h_part(grid.x,0);
    int *d_input=nullptr;
    int *d_part=nullptr;
    init(h_input);
    hipMalloc(reinterpret_cast<void**>(&d_input),input_byte_size);
    hipMalloc(reinterpret_cast<void**>(&d_part),part_byte_size);
    hipMemset(d_part,0,part_byte_size);
    hipMemcpy(d_input,h_input.data(),input_byte_size,hipMemcpyHostToDevice);
    kernel<<<grid,blocks>>>(d_input,d_part,block_size);
    hipDeviceSynchronize();
    hipMemcpy(h_part.data(),d_part,part_byte_size,hipMemcpyDeviceToHost);
    hipFree(d_part),hipFree(d_input);
    hipDeviceReset();
    seq_array_accum(res,h_part);
    std::cout<<res<<std::endl;
    return 0;
}