#include<bits/stdc++.h>
#include<hip/hip_runtime.h>

void init(std::vector<int>&arr)
{
    for (int i=0;i<(int)arr.size();++i)
    {
        arr[i]=i%10;
    }
}
__global__ void unrolling_2_blocks(int *input, int *part,int size)
{
    auto tid=threadIdx.x;
    auto block_offset=blockIdx.x*blockDim.x*2;
    auto index=block_offset+tid;
    int *window=input+block_offset;
    if ((index+blockDim.x)<size)
    {
        input[index]+=input[index+blockDim.x];
    }
    __syncthreads();
    for (int offset=blockDim.x/2;offset>0;offset=offset/2)
    {
        if (tid<offset)
        {
            window[tid]+=window[tid+offset];
        }
        __syncthreads();
    }
    if (tid==0)
    {
        part[blockIdx.x]=window[0];
    }
    __syncthreads();

}
__global__ void unrolling_4_blocks(int *input, int *part,int size)
{
    auto tid=threadIdx.x;
    auto block_offset=blockIdx.x*blockDim.x*4;
    auto index=block_offset+tid;
    int *window=input+block_offset;
    if ((index+3*blockDim.x)<size)
    {
        int a1=input[index];
        int a2=input[index+blockDim.x];
        int a3=input[index+2*blockDim.x];
        int a4=input[index+3*blockDim.x];

        input[index]=a1+a2+a3+a4;
    }
    __syncthreads();
    for (auto offset=blockDim.x/2;offset>0;offset/=2)
    {
        if (tid<offset)
        {
            window[tid]+=window[tid+offset];
        }
        __syncthreads();
    }
    if (tid==0)
    {
        part[blockIdx.x]=window[0];
    }

}
void seq_array_accum(int &a, const std::vector<int> &arr)
{
    for (const auto &i : arr)
    {
        a+=i;
    }
}
void run_unrolling_2blocks_kernel()
{
    int size=1<<27;
    int block_size=128;
    int cpu_res=0,gpu_res=0;
    dim3 blocks(block_size);
    dim3 grid((size/block_size)/2);
    const auto input_byte_size=size*sizeof(int);
    const auto part_byte_size=grid.x*sizeof(int);
    std::vector<int> h_input(size);
    std::vector<int>h_part(grid.x,0);
    int *d_input=nullptr;
    int *d_part=nullptr;
    init(h_input);
    seq_array_accum(cpu_res,h_input);
    hipMalloc(reinterpret_cast<void**>(&d_input),input_byte_size);
    hipMalloc(reinterpret_cast<void**>(&d_part),part_byte_size);
    hipMemset(d_part,0,part_byte_size);
    hipMemcpy(d_input,h_input.data(),input_byte_size,hipMemcpyHostToDevice);
    unrolling_2_blocks<<<grid,blocks>>>(d_input,d_part,size);
    hipDeviceSynchronize();
    hipMemcpy(h_part.data(),d_part,part_byte_size,hipMemcpyDeviceToHost);
    seq_array_accum(gpu_res,h_part);
    std::cout<<gpu_res<<" "<<cpu_res<<"\n";
    std::cout<<(gpu_res==cpu_res)<<"\n";
    h_input.clear(),h_part.clear(),hipFree(d_part),hipFree(d_input);
    hipDeviceReset();

}
void run_unrolling_4blocks_kernel()
{
    int size=1<<27;
    int block_size=128;
    int cpu_res=0,gpu_res=0;
    dim3 blocks(block_size);
    dim3 grid((size/block_size)/4);
    const auto input_byte_size=size*sizeof(int);
    const auto part_byte_size=grid.x*sizeof(int);
    std::vector<int> h_input(size);
    std::vector<int>h_part(grid.x,0);
    int *d_input=nullptr;
    int *d_part=nullptr;
    init(h_input);
    seq_array_accum(cpu_res,h_input);
    hipMalloc(reinterpret_cast<void**>(&d_input),input_byte_size);
    hipMalloc(reinterpret_cast<void**>(&d_part),part_byte_size);
    hipMemset(d_part,0,part_byte_size);
    hipMemcpy(d_input,h_input.data(),input_byte_size,hipMemcpyHostToDevice);
    unrolling_4_blocks<<<grid,blocks>>>(d_input,d_part,size);
    hipDeviceSynchronize();
    hipMemcpy(h_part.data(),d_part,part_byte_size,hipMemcpyDeviceToHost);
    seq_array_accum(gpu_res,h_part);
    std::cout<<gpu_res<<" "<<cpu_res<<"\n";
    std::cout<<(gpu_res==cpu_res)<<"\n";
    h_input.clear(),h_part.clear(),hipFree(d_part),hipFree(d_input);
    hipDeviceReset();
}
int main()
{
    run_unrolling_2blocks_kernel();
    run_unrolling_4blocks_kernel();
    return 0;
}