#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

#define BDMX 32
#define BDMY 32

const int bytes=sizeof(int)*BDMX*BDMY,nx=BDMX,ny=BDMY;
dim3 grid (1,1),blocks(nx,ny);
__global__ void kernel_1(int *out){
    __shared__ int sm[BDMY][BDMX];
    int tidx=threadIdx.x,tidy=threadIdx.y;
    int tgid=tidx+tidy*blockDim.x;
    sm[tidy][tidx]=tgid;
    __syncthreads();
    out[tgid]=sm[tidx][tidy];
}
__global__ void kernel_2(int *out){
    __shared__ int sm[BDMY][BDMX];
    int tidx=threadIdx.x,tidy=threadIdx.y;
    int tgid=tidx+tidy*blockDim.x;
    sm[tidx][tidy]=tgid;
    __syncthreads();
    out[tgid]=sm[tidy][tidx];
}
__global__ void kernel_3(int *out){
    __shared__ int sm[BDMY][BDMX];
    int tidx=threadIdx.x,tidy=threadIdx.y;
    int tgid=tidx+tidy*blockDim.x;
    sm[tidy][tidx]=tgid;
    __syncthreads();
    out[tgid]=sm[tidy][tidx];
}
void run_read_row_store_col_kernel(hipSharedMemConfig sm_config)
{
    hipDeviceSetSharedMemConfig(sm_config);
    std::vector<int>h_a(nx*ny);
    int *d_a= nullptr;
    hipMalloc(reinterpret_cast<void **>(&d_a),bytes);
    hipMemset(d_a,0,bytes);
    kernel_1<<<grid,blocks>>>(d_a);
    hipDeviceSynchronize();
    hipMemcpy(h_a.data(),d_a,bytes,hipMemcpyDeviceToHost);
    hipFree(d_a);
    h_a.clear();
    hipDeviceReset();
}
void run_read_col_store_row_kernel(hipSharedMemConfig sm_config)
{
    hipDeviceSetSharedMemConfig(sm_config);
    std::vector<int>h_a(nx*ny);
    int *d_a= nullptr;
    hipMalloc(reinterpret_cast<void **>(&d_a),bytes);
    hipMemset(d_a,0,bytes);
    kernel_2<<<grid,blocks>>>(d_a);
    hipDeviceSynchronize();
    hipMemcpy(h_a.data(),d_a,bytes,hipMemcpyDeviceToHost);
    hipFree(d_a);
    h_a.clear();
    hipDeviceReset();
}
void run_read_row_store_row_kernel(hipSharedMemConfig sm_config)
{
    hipDeviceSetSharedMemConfig(sm_config);
    std::vector<int>h_a(nx*ny);
    int *d_a= nullptr;
    hipMalloc(reinterpret_cast<void **>(&d_a),bytes);
    hipMemset(d_a,0,bytes);
    kernel_3<<<grid,blocks>>>(d_a);
    hipDeviceSynchronize();
    hipMemcpy(h_a.data(),d_a,bytes,hipMemcpyDeviceToHost);
    hipFree(d_a);
    h_a.clear();
    hipDeviceReset();
}
int main() {
    hipSharedMemConfig sm_32(hipSharedMemBankSizeFourByte);
    hipSharedMemConfig sm_64(hipSharedMemBankSizeEightByte);
    run_read_col_store_row_kernel(sm_32);
    run_read_row_store_col_kernel(sm_32);
    run_read_row_store_row_kernel(sm_32);
    run_read_col_store_row_kernel(sm_64);
    run_read_row_store_col_kernel(sm_64);
    run_read_row_store_row_kernel(sm_64);

    return 0;
}
