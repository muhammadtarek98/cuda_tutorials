#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
__global__ void kernel (float *a,float *b,float *c,int sz)
{
    int tgid=threadIdx.x+blockDim.x*blockIdx.x;
    if (tgid<sz)
    {
        c[tgid]=a[tgid]+b[tgid];
    }
}
void init(float *arr,int sz)
{
    for (int i=0;i<sz;++i)
    {
        arr[i]=i%10;
    }
}
void seq_array_sum(const float *a, const float *b, float *c,int sz)
{
    for (int i=0;i<sz;++i)
    {
        c[i]+=a[i]+b[i];
    }
}
int main()
{
    int sz=1<<22,block_sz=128;
    dim3 blocks(block_sz);
    dim3 grid((sz+blocks.x-1)/blocks.x);
    auto bytes=sizeof(float)*sz;
    float *a,*b,*c,*ref;
    c=(float*)malloc(bytes);
    hipMallocManaged(reinterpret_cast<void**>(&a),bytes);
    hipMallocManaged(reinterpret_cast<void**>(&b),bytes);
    hipMallocManaged(reinterpret_cast<void**>(&ref),bytes);
    init(a,sz),init(b,sz);
    seq_array_sum(a,b,c,sz);
    kernel<<<grid,blocks>>>(a,b,ref,sz);
    hipDeviceSynchronize();
    free(c);
    return 0;
}