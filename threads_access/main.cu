
#include <hip/hip_runtime.h>
#include <iostream>
__global__ void print_thread_idx()
{
    printf("threadIdx.x:%d threadIdx.y:%d threadIdx.z:%d\n",threadIdx.x,threadIdx.y,threadIdx.z);
}
int main()
{
    int nx=16,ny=16;
    dim3 block(8,8),grid(nx/block.x,ny/block.y);
    print_thread_idx<<<grid,block>>>();
    hipDeviceSynchronize();
    hipDeviceReset();

    return 0;
}