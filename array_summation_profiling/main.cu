#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

__global__ void sum_arrays_1Dgrid_1Dblock(float *a,float *b,float *c,int nx)
{
    auto tid=threadIdx.x+blockIdx.x;
    c[tid]=a[tid]+b[tid];
}
__global__ void sum_arrays_2Dgrid_2Dblock(float *a,float *b,float *c,int nx,int ny)
{
    auto tidx=blockIdx.x*blockDim.x+threadIdx.x;
    auto tidy=blockDim.y*blockIdx.y+threadIdx.y;
    auto tid=tidy*nx+tidx;
    if (tid<nx&&tid<ny)
        c[tid]=a[tid]+b[tid];

}
void sum_array_cpu(const float *a,const float *b,float *c,const int &size){
for (size_t i=0;i<size;++i)
{
    c[i]=a[i]+b[i];
}
}
void run_sum_array_1d(int argc,char const *argv[])
{
    int size=1<<22;
    int block_size=128;
    if (argc>2)
    {
        size=1<<atoi(argv[2]);
    }
    if (argc>4)
    {
        block_size=1<<atoi(argv[4]);
    }
    auto byte_size=size*sizeof(float);
    float *h_a,*h_b,*h_c;
    float *d_a,*d_b,*d_c;
    h_a=(float*)malloc(byte_size);
    h_b=(float*)malloc(byte_size);
    h_c=(float*)malloc(byte_size);
    if (!h_a)
    {
        std::cout<<"host allocation error\n";
    }
    for (size_t i =0;i<size;++i)
    {
        h_a[i]=i%10;
        h_b[i]=i%7;
    }
    sum_array_cpu(h_a,h_b,h_c,size);
    dim3 block(block_size);
    dim3 grid((size+block.x-1)/block.x);
    hipMalloc(reinterpret_cast<void**>(&d_a),byte_size);
    hipMalloc(reinterpret_cast<void**>(&d_b),byte_size);
    hipMalloc(reinterpret_cast<void**>(&d_c),byte_size);
    hipMemset(d_c,0,byte_size);
    hipMemcpy(d_a,h_a,byte_size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,byte_size,hipMemcpyHostToDevice);
    sum_arrays_1Dgrid_1Dblock<<<grid,block_size>>>(h_a,h_b,h_c,size);
    hipDeviceSynchronize();
    hipMemcpy(h_c,d_c,byte_size,hipMemcpyDeviceToHost);
    hipFree(d_a),hipFree(d_b),hipFree(d_c);
    free(h_a),free(h_b),free(h_c);

}
void run_sum_array_2d(int argc,char const *argv[])
{
    int size=1<<22;
    int block_x=128;
int block_y=8;
    int nx=1<<14;
    int ny=size/nx;
    if (argc>4)
    {
        int pow=atoi(argv[4]);
        if (pow<3||pow>10)
        {
            std::cout<<"invalid configs \n";
        }
        else
        {
            block_x=1<<pow;
            block_y=1024/block_x;
        }
    }
    auto bytsize=size*sizeof(float);
    float *h_a,*h_b,*h_c;

    h_a=static_cast<float*>(malloc(bytsize));
    h_b=static_cast<float*>(malloc(bytsize));
    h_c=static_cast<float*>(malloc(bytsize));
    memset(h_c,0,bytsize);
    for (size_t i =0;i<size;++i)
    {
        h_a[i]=i%10;
        h_b[i]=i%7;
    }
    dim3 block_size(block_x,block_y);
    dim3 grid_size((nx+block_x-1)/block_x,(ny+block_y-1)/block_y);
    sum_array_cpu(h_a,h_b,h_c,size);
    float *d_a,*d_b,*d_c;
    hipMalloc(reinterpret_cast<void**>(&d_a),bytsize);
    hipMalloc(reinterpret_cast<void**>(&d_b),bytsize);
    hipMalloc(reinterpret_cast<void**>(&d_c),bytsize);
    hipMemset(d_c,0,bytsize);
    hipMemcpy(d_a,h_a,bytsize,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,bytsize,hipMemcpyHostToDevice);
    sum_arrays_2Dgrid_2Dblock<<<grid_size,block_size>>>(d_a,d_b,d_c,nx,ny);
    hipDeviceSynchronize();
    hipMemcpy(h_c,d_c,bytsize,hipMemcpyDeviceToHost);
    hipFree(d_a),hipFree(d_b),hipFree(d_c);
    free(h_a),free(h_b),free(h_c);

}
int main(const int argc,char const *argv[])
{
    if (argc>1)
    {
        if (atoi(argv[1])>0)
        {
            run_sum_array_2d(argc,argv);
        }
        else
        {
            run_sum_array_1d(argc,argv);
        }
    }
    else
    {
        run_sum_array_1d(argc,argv);
    }


    return 0;
}