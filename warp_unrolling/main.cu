#include<bits/stdc++.h>

#include <hip/hip_runtime.h>
 __global__ void warp_unrolling(int *input,int *part,int size){
    auto tid=threadIdx.x;
     auto idx=blockDim.x*blockIdx.x+tid;
     int *window=input+blockDim.x*blockIdx.x;
     for (auto offset=blockDim.x/2;offset>=64;offset/=2)
     {
         if (tid<offset)
         {
             window[tid]+=window[tid+offset];
         }
         __syncthreads();
     }
     if (tid<32)
     {
         volatile int *vsmem=window;
         vsmem[tid]+=vsmem[tid+32];
         vsmem[tid]+=vsmem[tid+16];
         vsmem[tid]+=vsmem[tid+8];
         vsmem[tid]+=vsmem[tid+4];
         vsmem[tid]+=vsmem[tid+2];
         vsmem[tid]+=vsmem[tid+1];
     }
     if (tid==0)
     {
         part[blockIdx.x]=window[0];
     }



 }
void seq_array_accum(int &a, const std::vector<int> &arr)
 {
     for (const auto &i : arr)
     {
         a+=i;
     }
 }
void init(std::vector<int>&arr)
 {
     for (int i=0;i<(int)arr.size();++i)
     {
         arr[i]=i%10;
     }
 }
int main(int argc,const char *argv [])
{
     int size=1<<27;
     int block_size=128;
     int cpu_res=0,gpu_res=0;
     dim3 blocks(block_size);
     dim3 grid((size/block_size));
     const auto input_byte_size=size*sizeof(int);
     const auto part_byte_size=grid.x*sizeof(int);
     std::vector<int> h_input(size);
     std::vector<int>h_part(grid.x,0);
     int *d_input=nullptr;
     int *d_part=nullptr;
     init(h_input);
     seq_array_accum(cpu_res,h_input);
     hipMalloc(reinterpret_cast<void**>(&d_input),input_byte_size);
     hipMalloc(reinterpret_cast<void**>(&d_part),part_byte_size);
     hipMemset(d_part,0,part_byte_size);
     hipMemcpy(d_input,h_input.data(),input_byte_size,hipMemcpyHostToDevice);
     warp_unrolling<<<grid,blocks>>>(d_input,d_part,size);
     hipDeviceSynchronize();
     hipMemcpy(h_part.data(),d_part,part_byte_size,hipMemcpyDeviceToHost);
     seq_array_accum(gpu_res,h_part);
     std::cout<<gpu_res<<" "<<cpu_res<<"\n";
     std::cout<<(gpu_res==cpu_res)<<"\n";
     h_input.clear(),h_part.clear(),hipFree(d_part),hipFree(d_input);
     hipDeviceReset();
    return 0;
}