#include<bits/stdc++.h>

#include <hip/hip_runtime.h>
__global__ void  kernel(int *res,int size){
    auto tgid=threadIdx.x+blockDim.x*blockIdx.x;
    int x1=21,x2=45,x3=54;
    int x4=x1+x2+x3;
    if(tgid<size){
        res[tgid]=x4;
    }
    __syncthreads();

}
int main(int argc,const char *argv [])
{
    /*
     compile with nvcc --ptxas-options=-v -o main.out main.cu
     */
    int block_size=128;
    int size=1<<22;
     dim3 block(block_size);
    int byte_size=sizeof (int)*size;
    int *h_res=(int*)(malloc(byte_size));
    int *d_res=(int*)(malloc(byte_size));
    hipMalloc(reinterpret_cast<void **>(&d_res),byte_size);
    hipMemset(d_res,0,byte_size);
     dim3 grid((size+block.x-1)/block.x);
    kernel<<<grid,block>>>(d_res,size);
    hipDeviceSynchronize();
    hipMemcpy(h_res,d_res,byte_size,hipMemcpyDeviceToHost);
    hipFree(h_res);
    std::cout<<*h_res;
    delete h_res;

    return 0;
}