#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

__global__ void kernel()
{
    printf( "Hello, World!\n");

}
int main()
{
    int nx=16,ny=4;
    dim3 block(nx/2,ny/2);
    dim3 grid(nx/block.x,ny/block.y);
    kernel<<<grid,block  >>>();
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}