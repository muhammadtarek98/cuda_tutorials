#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
const int block_size=2;
struct Configs
{
    dim3 dimsA,dimsB,dimsC;
    int mem_size_A,mem_size_B,mem_size_C;
    int size_a,size_b,size_c;
    int h2d,d2h;
    dim3 threads,grid;
    Configs()
    {
        this->dimsA=dim3(5 * 2 * block_size, 5 * 2 * block_size, 1);
        this->dimsB=dim3(5 * 4 * block_size, 5 * 2 * block_size, 1);

        this->dimsC=dim3(this->dimsB.x, this->dimsA.y, 1);
        this->size_a=static_cast<int>(this->dimsA.x * this->dimsA.y);
        this->size_b=static_cast<int>(this->dimsB.x * this->dimsB.y);
        this->size_c=static_cast<int>(this->dimsC.x * this->dimsC.y);
        this->mem_size_A=static_cast<int>(this->size_a*sizeof(int));
        this->mem_size_B=static_cast<int>(this->size_b*sizeof(int));
        this->mem_size_C=static_cast<int>(this->size_c*sizeof(int));
        this->h2d=hipMemcpyHostToDevice;
        this->d2h=hipMemcpyDeviceToHost;
        this->threads=dim3(block_size, block_size);
        this->grid=dim3(this->dimsB.x / this->threads.x, this->dimsA.y / this->threads.y);
    }
};

__global__ void MatMul(int *A,int*B,int*C, int xA, int xB)
{
    auto bx=blockIdx.x;
    auto by=blockIdx.y;
    auto tx=threadIdx.x;
    auto ty=threadIdx.y;
    auto a_begin=xA*block_size*by;
    auto a_end=a_begin+xA-1;
    auto b_begin=xB*block_size;
    auto a_step=block_size,b_step=block_size*xB;
    int C_sub=0;

    for (int a=static_cast<int>(a_begin),b=(b_begin);a<=a_end;a+=a_step,b+=b_step)
    {
        __shared__ int As[block_size][block_size];
        __shared__ int Bs[block_size][block_size];
        As[ty][tx]=A[a+xA*ty+tx];
        Bs[ty][tx]=B[b+xB*ty+tx];
        __syncthreads();
        for (int k=0;k<block_size;++k)
        {
            C_sub+=As[ty][k]*Bs[k][tx];
        }
        __syncthreads();

    }

    auto c_step=xB*block_size*by+block_size*bx;
    C[c_step+xB*ty+tx]=C_sub;
}
void print(int* arr, int size) {
    for (int i = 0; i < size; i++) {
        std::cout << arr[i] << " ";
    }
    std::cout << std::endl;
}
void random_ints(int* arr, int size) {
    srand(time(NULL));
    for(int i = 0; i < size; i++) {
        arr[i] = int(rand() % 10);
    }
}
int main()
{
    int devID=0;
    hipSetDevice(devID);
    Configs config;
std::cout << "dimsA: " << config.dimsA.x << " x " << config.dimsA.y << " x " << config.dimsA.z << "\n";
std::cout << "dimsB: " << config.dimsB.x << " x " << config.dimsB.y << " x " << config.dimsB.z << "\n";
    std::cout << "dimsC: " << config.dimsC.x << " x " << config.dimsC.y << " x " << config.dimsC.z << "\n";

    int*matA =static_cast<int*>(malloc(config.mem_size_A));
    int*matB =static_cast<int*>(malloc(config.mem_size_B));
    int*matC=static_cast<int*>(malloc(config.mem_size_C));
    random_ints(matA, config.size_a);
    random_ints(matB, config.size_b);
    print(matA,config.size_a);
    print(matB,config.size_b);
    int *d_matA=nullptr,*d_matB=nullptr,*d_matC=nullptr;
    hipMalloc((void**)&d_matA,config.mem_size_A);
    hipMalloc((void**)&d_matB,config.mem_size_B);
    hipMalloc((void**)&d_matC,config.mem_size_C);
    hipMemcpy(d_matA,matA,config.mem_size_A,static_cast<hipMemcpyKind>(config.h2d));
    hipMemcpy(d_matB,matB,config.mem_size_B,static_cast<hipMemcpyKind>(config.h2d));
    MatMul<<<config.grid,config.threads>>>(d_matA,d_matB,d_matC,static_cast<int>(config.dimsA.x),static_cast<int>(config.dimsB.x));
    hipMemcpy(matC,d_matC,config.mem_size_C,static_cast<hipMemcpyKind>(config.d2h));
    print(matC,config.size_c);
    free(matA),free(matB),free(matC);
    hipFree(d_matA),hipFree(d_matB),hipFree(d_matC);

    return 0;

}