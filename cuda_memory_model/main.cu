#include<bits/stdc++.h>

#include <hip/hip_runtime.h>
__global__ void kernel(int *a,int *b,int *c,const int size)
{
    int tgid=threadIdx.x+blockIdx.x*blockDim.x;
    if (tgid<size)
    {
        c[tgid]=a[tgid]+b[tgid];
    }
    __syncthreads();
}
void init(std::vector<int>&arr)
{
    for (int i=0;i<(int)arr.size();++i)
    {
        arr[i]=i%10;
    }
}
int main(int argc,const char *argv [])
{
    int size=1<<22,block_size=128;
    dim3 blocksize(block_size);
    dim3 grid((size+block_size-1)/block_size);
    if (argc>1)
    {
        block_size=1<<atoi(argv[1]);
    }
    auto byte_size=size*sizeof(int);
    std::vector<int>a(size);
    std::vector<int>b(size);
    std::vector<int>c(size,0);
    init(a),init(b);

    int *d_a=nullptr,*d_b=nullptr,*d_c=nullptr;
    hipMalloc(reinterpret_cast<void**>(&d_a),byte_size);
    hipMalloc(reinterpret_cast<void**>(&d_b),byte_size);
    hipMalloc(reinterpret_cast<void**>(&d_c),byte_size);
    hipMemset(d_c,0,byte_size);
    hipMemcpy(d_a,a.data(),byte_size,hipMemcpyHostToDevice);
    hipMemcpy(d_c,b.data(),byte_size,hipMemcpyHostToDevice);
    kernel<<<grid,blocksize>>>(d_a,d_b,d_c,size);
    hipDeviceSynchronize();
    hipMemcpy(c.data(),d_c,byte_size,hipMemcpyDeviceToHost);
    hipFree(d_a),hipFree(d_c),hipFree(d_b);
    a.clear();
    b.clear();
    c.clear();

    return 0;
}